/*
The code is modfied from
https://github.com/state-spaces/mamba
*/

// Split into multiple files to compile in paralell
#include "quant_sscan_update_kernel.cuh"

// quant_sscan_fwd_cuda<input_t, weight_t>(params, stream)
// int8_t -> symmetric quant; uint8_t -> asymmetric quant 
template void quant_sscan_update_cuda<int8_t, int8_t>(QuantSSMParams &params, hipStream_t stream);
// void quant_sscan_update_cuda(int8_t *u);